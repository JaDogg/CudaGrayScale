#include "hip/hip_runtime.h"
/*************************************************
 *	Part of cuda bitmap to grayscale converter   *
 *	- Bhathiya Perera                            *
 *************************************************/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "utils.cuh"
#include "bitmap.cuh"
#define PIXEL_SIZE 3

// My GPU Has 1024 Threads per block, thus 32x32 threads
// 32 = 2^5, therefore 32 is 5 bits
#define THREAD_PER_2D_BLOCK 32
#define THREAD_PER_2D_BLOCK_BITS 5

hipError_t turnGrayWithCuda(unsigned char* bitmapData, BitmapInfoHeader* header, unsigned int size);

// Turn given bitmap data to gray scale
__global__ void turnGray(unsigned char* bitmapData, unsigned long size, unsigned int width)
{
	// This is done because shifting left by 5 is faster than multiplying by 32
#define xIndex ((blockIdx.x << THREAD_PER_2D_BLOCK_BITS) + threadIdx.x)
#define yIndex ((blockIdx.y << THREAD_PER_2D_BLOCK_BITS) + threadIdx.y)
#define BLUE bitmapData[dataIndex]
#define GREEN bitmapData[dataIndex+1]
#define RED bitmapData[dataIndex+2]
	unsigned long dataIndex = (xIndex + (yIndex * width)) * PIXEL_SIZE;
	// Gray occurs when RED == GREEN == BLUE, so get average
	if(dataIndex < size) {
		// This is done because shifting right is faster than division
		// And average can be calculated in two steps
		unsigned char gray = (((RED + GREEN) >> 1) + BLUE) >> 1;
		// Convert all pixels to gray
		RED = gray;
		GREEN = gray;
		BLUE = gray;
	}
#undef RED
#undef GREEN
#undef BLUE
#undef yIndex
#undef xIndex
}

void printHelp(char* binary)
{
	printf("GrayScaleCUDA\n");
	printf("----------------------------------");
	printf("\t-Bhathiya Perera\n");
	printf("Execute: %s <Bitmap>\n", binary);
}

int main(int argc, char** argv)
{
// Freeing data and calling hipDeviceReset must be done
// All the time
#undef DO_FAILED_EXIT
#define DO_FAILED_EXIT()\
	free(header);\
	free(data);\
	hipDeviceReset();\
	return EXIT_FAILURE;

	if (argc != 2) {
		printHelp(argv[0]);
		return EXIT_FAILURE;
	}

#ifdef DEBUG
#define bitmapFilename "C:\\Users\\Bhathiya\\Desktop\\img.bmp"
#else
#define bitmapFilename argv[1]
#endif

	puts("--------------------------------------------------");
	LOG("Welcome to grayscale with CUDA.");
	LOG("Turning %s to grayscale...", bitmapFilename);

	BitmapInfoHeader* header = 0;
	header = (BitmapInfoHeader*)malloc(sizeof(BitmapInfoHeader));
	unsigned char* data = loadBitmapFile(bitmapFilename, header);
	if (data==NULL) {
		LOG_ERROR("Failed to load bitmap");
		DO_FAILED_EXIT();
	}

	hipError_t cudaStatus = turnGrayWithCuda(data, header, header->sizeImage);
    REPORT_CUDA_ERROR(cudaStatus, "Unable to turn grayscale with cuda");

	int success = overwriteBitmapData(bitmapFilename, data);
	if(!success) {
		LOG_ERROR("Failed to overwrite bitmap");
		DO_FAILED_EXIT();
	}

	free(header);
	free(data);
	hipDeviceReset();
    return EXIT_SUCCESS;
}

// Helper function for using CUDA to convert bitmap data to gray
hipError_t turnGrayWithCuda(unsigned char* bitmapData, BitmapInfoHeader* header, unsigned int size)
{
#undef DO_FAILED_EXIT
#define DO_FAILED_EXIT() hipFree(devBitmap); return cudaStatus;
	unsigned char* devBitmap = 0;
    hipError_t cudaStatus;
	size_t dataSize = size * sizeof(unsigned char);
	unsigned long pixelCount = size / PIXEL_SIZE;
	LOG("size=%d, dataSize=%d, pixelCount=%d", size, dataSize, pixelCount);
	LOG("Image Width=%d Height=%d", header->width, header->height);
	cudaStatus = selectBestDevice();
	REPORT_CUDA_ERROR(cudaStatus, "Unable to select a cuda device! "
		"Do you have a CUDA-capable GPU installed?");

	// Allocate GPU buffer for bitmap data
    cudaStatus = hipMalloc((void**)&devBitmap, dataSize);
	REPORT_CUDA_ERROR(cudaStatus, "Unable allocate device memory");

	// Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(devBitmap, bitmapData, dataSize,
		hipMemcpyHostToDevice);
	REPORT_CUDA_ERROR(cudaStatus, "Copying memory failed!");

	// Calculate number of threadsPerBlock and blocksPerGrid
	dim3 threadsPerBlock(THREAD_PER_2D_BLOCK, THREAD_PER_2D_BLOCK);
	// Need to consider integer devision, and It's lack of precision
	// This way total number of threads are newer lower than pixelCount
	dim3 blocksPerGrid((header->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(header->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    LOG("CUDA kernel launch with %dx%d blocks of %dx%d threads. Total threads=%d",
		blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y,
		blocksPerGrid.x * blocksPerGrid.y * threadsPerBlock.x * threadsPerBlock.y);

	CUDA_START_TIMER;
	// Launch a kernel on the GPU
	turnGray<<<blocksPerGrid, threadsPerBlock>>>(devBitmap, size, header->width);
	CUDA_STOP_TIMER;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
	REPORT_CUDA_ERROR(cudaStatus, "Kernel launch failed: %s",
		hipGetErrorString(cudaStatus));

    // Function hipDeviceSynchronize waits for the kernel to finish, and returns
    // Any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
	REPORT_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize() returned error"
		" code %d after launching kernel!", cudaStatus);

	// Log Effective Bandwidth and total time
	// It is necessary to multiply by 2 because both read and write operations
	// Occur
	CUDA_LOG_TIME(size * 2.0f / milliseconds / 1e6f);

    // Copy bitmap data from GPU buffer to host memory.
    cudaStatus = hipMemcpy(bitmapData, devBitmap, dataSize,
		hipMemcpyDeviceToHost);
	REPORT_CUDA_ERROR(cudaStatus, "Copying memory failed!");

    hipFree(devBitmap);
    return cudaStatus;
}